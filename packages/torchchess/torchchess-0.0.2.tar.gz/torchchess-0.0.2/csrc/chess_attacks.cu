#include "hip/hip_runtime.h"
#pragma once
#include <torch/extension.h>
#include "chess_consts.h"
#include "clamp.cu"

__device__ unsigned char count_attacks(
    size_t env, unsigned char row, unsigned char col, 
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> players ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> boards
) {
    long attacks = 0;

    // relative pieces
    const unsigned char enemy_knight = ((players[env] + 1) % 2 * 6) + WHITE_KNIGHT;
    const unsigned char enemy_bishop = ((players[env] + 1) % 2 * 6) + WHITE_BISHOP;
    const unsigned char enemy_rook   = ((players[env] + 1) % 2 * 6) + WHITE_ROOK;
    const unsigned char enemy_queen  = ((players[env] + 1) % 2 * 6) + WHITE_QUEEN;
    const unsigned char enemy_king   = ((players[env] + 1) % 2 * 6) + WHITE_KING;

    // if player is white count attacks by blacks pawns
    attacks += (players[env] == WHITE) & (row > 0) & (col > 0) & (boards[env][clamp(0,63,(row - 1) * 8 + col - 1)] == BLACK_PAWN);
    attacks += (players[env] == WHITE) & (row > 0) & (col < 7) & (boards[env][clamp(0,63,(row - 1) * 8 + col + 1)] == BLACK_PAWN);

    // if player is black count attacks by white pawns
    attacks += (players[env] == BLACK) & (row < 7) & (col > 0) & (boards[env][clamp(0,63,(row + 1) * 8 + col - 1)] == WHITE_PAWN);
    attacks += (players[env] == BLACK) & (row < 7) & (col < 7) & (boards[env][clamp(0,63,(row + 1) * 8 + col + 1)] == WHITE_PAWN);
    
    // count knight attacks
    attacks += (row > 1) & (col > 0) & (boards[env][clamp(0,63,(row - 2) * 8 + (col - 1))] == enemy_knight);
    attacks += (row > 1) & (col < 7) & (boards[env][clamp(0,63,(row - 2) * 8 + (col + 1))] == enemy_knight);
    attacks += (row > 0) & (col > 1) & (boards[env][clamp(0,63,(row - 1) * 8 + (col - 2))] == enemy_knight);
    attacks += (row > 0) & (col < 6) & (boards[env][clamp(0,63,(row - 1) * 8 + (col + 2))] == enemy_knight);
    attacks += (row < 7) & (col > 1) & (boards[env][clamp(0,63,(row + 1) * 8 + (col - 2))] == enemy_knight);
    attacks += (row < 7) & (col < 6) & (boards[env][clamp(0,63,(row + 1) * 8 + (col + 2))] == enemy_knight);
    attacks += (row < 6) & (col > 0) & (boards[env][clamp(0,63,(row + 2) * 8 + (col - 1))] == enemy_knight);
    attacks += (row < 6) & (col < 7) & (boards[env][clamp(0,63,(row + 2) * 8 + (col + 1))] == enemy_knight);
    
    // count king attacks
    attacks += (row > 0) & (col > 0) & (boards[env][clamp(0,63,(row - 1) * 8 + (col - 1))] == enemy_king);
    attacks += (row > 0) & (col < 7) & (boards[env][clamp(0,63,(row - 1) * 8 + (col + 1))] == enemy_king);
    attacks += (row < 7) & (col > 0) & (boards[env][clamp(0,63,(row + 1) * 8 + (col - 1))] == enemy_king);
    attacks += (row < 7) & (col < 7) & (boards[env][clamp(0,63,(row + 1) * 8 + (col + 1))] == enemy_king);
    attacks += (row > 0) & (boards[env][clamp(0,63,(row - 1) * 8 + col)] == enemy_king);
    attacks += (row < 7) & (boards[env][clamp(0,63,(row + 1) * 8 + col)] == enemy_king);
    attacks += (col > 0) & (boards[env][clamp(0,63,row * 8 + (col - 1))] == enemy_king);
    attacks += (col < 7) & (boards[env][clamp(0,63,row * 8 + (col + 1))] == enemy_king);
    
    
    // count bottom-right attacks
    bool covered = false;
    for (int i = 1; i < 8; i++) {
        attacks += (!covered) & (row + i < 8) & (col + i < 8) & (boards[env][clamp(0,63,(row + i) * 8 + (col + i))] == enemy_bishop | boards[env][clamp(0,63,(row + i) * 8 + (col + i))] == enemy_queen);
        covered = covered | (boards[env][clamp(0,63,(row + i) * 8 + (col + i))] != EMPTY);
    }
    
    // count bottom-left attacks
    covered = false;
    for (int i = 1; i < 8; i++) {
        attacks += (!covered) & (row + i < 8) & (col - i >= 0) & (boards[env][clamp(0,63,(row + i) * 8 + (col - i))] == enemy_bishop | boards[env][clamp(0,63,(row + i) * 8 + (col - i))] == enemy_queen);
        covered = covered | (boards[env][clamp(0,63,(row + i) * 8 + (col - i))] != EMPTY);
    }

    // count top-right attacks
    covered = false;
    for (int i = 1; i < 8; i++) {
        attacks += (!covered) & (row - i >= 0) & (col + i < 8) & (boards[env][clamp(0,63,(row - i) * 8 + (col + i))] == enemy_bishop | boards[env][clamp(0,63,(row - i) * 8 + (col + i))] == enemy_queen);
        covered = covered | (boards[env][clamp(0,63,(row - i) * 8 + (col + i))] != EMPTY);
    }

    // count top-left attacks
    covered = false;
    for (int i = 1; i < 8; i++) {
        attacks += (!covered) & (row - i >= 0) & (col - i >= 0) & (boards[env][clamp(0,63,(row - i) * 8 + (col - i))] == enemy_bishop | boards[env][clamp(0,63,(row - i) * 8 + (col - i))] == enemy_queen);
        covered = covered | (boards[env][clamp(0,63,(row - i) * 8 + (col - i))] != EMPTY);
    }

    // count bottom attacks
    covered = false;
    for (int i = 1; i < 8; i++) {
        attacks += (!covered) & (row + i < 8) & (boards[env][clamp(0,63,(row + i) * 8 + col)] == enemy_rook | boards[env][clamp(0,63,(row + i) * 8 + col)] == enemy_queen);
        covered = covered | (boards[env][clamp(0,63,(row + i) * 8 + col)] != EMPTY);
    }

    // count top attacks
    covered = false;
    for (int i = 1; i < 8; i++) {
        attacks += (!covered) & (row - i >= 0) & (boards[env][clamp(0,63,(row - i) * 8 + col)] == enemy_rook | boards[env][clamp(0,63,(row - i) * 8 + col)] == enemy_queen);
        covered = covered | (boards[env][clamp(0,63,(row - i) * 8 + col)] != EMPTY);
    }

    // count right attacks
    covered = false;
    for (int i = 1; i < 8; i++) {
        attacks += (!covered) & (col + i < 8) & (boards[env][clamp(0,63,row * 8 + col + i)] == enemy_rook | boards[env][clamp(0,63,row * 8 + col + i)] == enemy_queen);
        covered = covered | (boards[env][clamp(0,63,row * 8 + col + i)] != EMPTY);
    }

    // count left attacks
    covered = false;
    for (int i = 1; i < 8; i++) {
        attacks += (!covered) & (col - i >= 0) & (boards[env][clamp(0,63,row * 8 + col - i)] == enemy_rook | boards[env][clamp(0,63,row * 8 + col - i)] == enemy_queen);
        covered = covered | (boards[env][clamp(0,63,row * 8 + col - i)] != EMPTY);
    }
    
    return attacks * (row <= 7 & col <= 7);

}

__global__ void attacks_kernel(
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> boards  ,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> players ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> colors
) {
    const size_t env = blockIdx.x;
    const unsigned char row = threadIdx.y;
    const unsigned char col = threadIdx.x;

    colors[env][row * 8 + col] = count_attacks(env, row, col, players, boards);
}
