#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <hip/hip_runtime.h>

#include <c10/cuda/CUDAStream.h>
#include <torch/types.h>

namespace {

template <typename T>
T getCudaDriverSymbol(const char* name) {
  void* fn = nullptr;
#if CUDA_VERSION >= 12000
  enum hipDriverEntryPointQueryResult queryResult;
  C10_CUDA_CHECK(
      hipGetDriverEntryPoint(name, &fn, hipEnableDefault, &queryResult));
  TORCH_CHECK(
      queryResult == hipDriverEntryPointSuccess,
      "Querying the ",
      name,
      " symbol from the CUDA driver failed with error ",
      queryResult);
#else // CUDA_VERSION < 12000
  C10_CUDA_CHECK(hipGetDriverEntryPoint(name, &fn, hipEnableDefault));
#endif // CUDA_VERSION
  TORCH_CHECK(
      fn != nullptr,
      "Querying the ",
      name,
      " symbol from the CUDA driver returned a null pointer");
  return reinterpret_cast<T>(fn);
}

void raiseCudaDriverError(hipError_t result, const char* fnName) {
  static PFN_cuGetErrorName myCuGetErrorName =
      getCudaDriverSymbol<PFN_cuGetErrorName>("hipDrvGetErrorName");
  static PFN_cuGetErrorString myCuGetErrorString =
      getCudaDriverSymbol<PFN_cuGetErrorString>("hipDrvGetErrorString");

  const char* ptr;
  hipError_t subResult = myCuGetErrorName(result, &ptr);
  std::string errorName = subResult == hipSuccess ? ptr : "UNKNOWN";
  subResult = myCuGetErrorString(result, &ptr);
  std::string errorString = subResult == hipSuccess ? ptr : "???";

  TORCH_CHECK(
      result == hipSuccess,
      "Calling ",
      fnName,
      " from the CUDA driver failed with error ",
      errorName,
      " (code ",
      result,
      "): ",
      errorString);
}

void cudaMemcpy32bAsync(
    torch::Tensor buffer,
    torch::Scalar value,
    torch::Stream stream) {
  static PFN_cuMemsetD32Async myCuMemsetD32Async =
      getCudaDriverSymbol<PFN_cuMemsetD32Async>("hipMemsetD32Async");

  TORCH_CHECK(buffer.is_cuda());
  TORCH_CHECK(buffer.dtype() == torch::kInt32);
  TORCH_CHECK(buffer.is_non_overlapping_and_dense());
  TORCH_CHECK(value.isIntegral(/*includeBool=*/false));
  hipError_t result = myCuMemsetD32Async(
      reinterpret_cast<hipDeviceptr_t>(buffer.data_ptr()),
      static_cast<unsigned int>(value.toInt()),
      buffer.numel(),
      c10::cuda::CUDAStream(stream).stream());
  if (result != hipSuccess) {
    raiseCudaDriverError(result, "hipMemsetD32Async");
  }
}

} // namespace

TORCH_LIBRARY_IMPL(xformers, CUDA, m) {
  m.impl(
      TORCH_SELECTIVE_NAME("xformers::cuda_memset_32b_async"),
      TORCH_FN(cudaMemcpy32bAsync));
}
