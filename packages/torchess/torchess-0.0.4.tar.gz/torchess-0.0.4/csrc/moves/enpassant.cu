#include "hip/hip_runtime.h"
#pragma once
#include <torch/extension.h>
#include "../chess_consts.h"

__device__ bool enpassant_move(
    size_t env,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> players ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> boards  ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> actions
) {
    // performs en passant
    // returns 0 if the action was performed
    // returns 1 if the action was not applicable
    
    const unsigned char player_pawn = players[env] * 6 + WHITE_PAWN;
    const unsigned char enemy_pawn  = ((players[env] + 1) % 2) * 6 + WHITE_PAWN;
    const unsigned char source = actions[env][0] * 8 + actions[env][1];
    const unsigned char target = actions[env][2] * 8 + actions[env][3];
    const unsigned char prev_action = WHITE_PREV1 + 10*((players[env]+1)%2);
    const unsigned char prev_target = boards[env][prev_action+2] * 8 + boards[env][prev_action+3];
    const unsigned char enpassant_src_row = players[env] == WHITE ? 3 : 4;
    const unsigned char enpassant_tgt_row = players[env] == WHITE ? 2 : 5;

    const bool is_action_ok = (
        (actions[env][4] == 0                                            ) & // no special action
        (actions[env][0] == enpassant_src_row                            ) & // action source is in en passant row
        (actions[env][2] == enpassant_tgt_row                            ) & // action target is in en passant row
        (abs(actions[env][1] - actions[env][3]) == 1                     ) & // moving on side column
        (boards[env][source] == player_pawn                              ) & // moving a pawn
        (boards[env][prev_action+4] == 0                                 ) & // previous action was a normal action 
        (boards[env][prev_action+3] == actions[env][3]                   ) & // previous action was a to the same column
        (abs(boards[env][prev_action] - boards[env][prev_action+2]) == 2 ) & // previous action was a double move
        (boards[env][prev_target] == enemy_pawn                          ) & // previous action moved a pawn
        (boards[env][target] == EMPTY                                    )   // action target is empty
    );

    boards[env][target] = is_action_ok ? player_pawn : boards[env][target];
    boards[env][source] = is_action_ok ? EMPTY       : boards[env][source];
    boards[env][prev_target] = is_action_ok ? EMPTY : boards[env][prev_target];

    return !is_action_ok;
}

__global__ void enpassant_kernel(
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> boards  ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> actions ,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> players ,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> result
) {
    const int env = blockIdx.x * blockDim.x + threadIdx.x;
    if (env < boards.size(0)) result[env] = enpassant_move(env, players, boards, actions);
}


