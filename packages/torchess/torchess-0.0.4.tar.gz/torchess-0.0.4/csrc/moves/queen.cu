#include "hip/hip_runtime.h"
#pragma once
#include <torch/extension.h>
#include "../chess_consts.h"
#include "../clamp.cu"

__device__ bool queen_move(
    size_t env,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> players ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> boards  ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> actions
) {
    // performs a queen movement
    // returns 0 if the action was performed
    // returns 1 if the action was not applicable
    // this routine does not verify if the queen is in check
    
    const unsigned char player_queen = players[env] * 6 + WHITE_QUEEN;
    const unsigned char source = actions[env][0] * 8 + actions[env][1];
    const unsigned char target = actions[env][2] * 8 + actions[env][3];
    const unsigned char enemy_pawn  = ((players[env] + 1) % 2) * 6 + WHITE_PAWN;
    const unsigned char enemy_queen = ((players[env] + 1) % 2) * 6 + WHITE_QUEEN;
    const unsigned char srcrow = actions[env][0];
    const unsigned char srccol = actions[env][1];
    const unsigned char tgtrow = actions[env][2];
    const unsigned char tgtcol = actions[env][3];

    const char dir_x = (+1) * (srccol < tgtcol) + (-1) * (srccol > tgtcol);
    const char dir_y = (+1) * (srcrow < tgtrow) + (-1) * (srcrow > tgtrow);
    bool is_jumping_over = false;
    bool encountered_target = false;
    for (int i = 1; i < 8; i++) {
        encountered_target = encountered_target | ((srcrow + i * dir_y == tgtrow) & (srccol + i * dir_x == tgtcol));
        is_jumping_over = is_jumping_over | ((!encountered_target) & (boards[env][clamp(0,63,(srcrow + i * dir_y) * 8 + (srccol + i * dir_x))] != EMPTY));
    }

    const bool is_action_ok = (
        (actions[env][4] == 0)                & // no special action
        (boards[env][source] == player_queen) & // source is a queen
        !is_jumping_over & (                    // queen is not jumping over other pieces
            ((srcrow == tgtrow) & (srccol <= 7)) |
            ((srccol == tgtcol) & (srcrow <= 7)) |
            (abs(srcrow - tgtrow) == abs(srccol - tgtcol))
        ) & ( // target is a valid queen movement
            (boards[env][target] == EMPTY) |
            ((boards[env][target] >= enemy_pawn) & (boards[env][target] <= enemy_queen))
        ) // target is empty or enemy
    );

    boards[env][target] = is_action_ok ? player_queen : boards[env][target];
    boards[env][source] = is_action_ok ? EMPTY       : boards[env][source];

    return !is_action_ok;
}

__global__ void queen_kernel(
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> boards  ,
    torch::PackedTensorAccessor32<int , 2 , torch::RestrictPtrTraits> actions ,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> players ,
    torch::PackedTensorAccessor32<int , 1 , torch::RestrictPtrTraits> result
) {
    const int env = blockIdx.x * blockDim.x + threadIdx.x;
    if (env < boards.size(0)) result[env] = queen_move(env, players, boards, actions);
}


